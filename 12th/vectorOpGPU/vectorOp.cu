#include "hip/hip_runtime.h"
#include "vectorOp.h"

__global__ void
vector_operation_kernel(int* output, int* data, int size)
{
  for (uint tid = blockDim.x * blockIdx.x + threadIdx.x; tid < size;
       tid += blockIdx.x * gridDim.x)
    output[tid] = OPERATION_I(data[tid]);
}

void
sequential_vector_operation(int* output, int* data, int size)
{
  for (int i = 0; i < size; i++) {
    output[i] = OPERATION_I(data[i]);
  }
  return;
}

int
main(int argc, char* argv[])
{
  double elapsed_time_CUDA, elapsed_time_CPU;
  int block_size, grid_size;
  int data_size;
  int *data_h, *output_h, *device_output_h;
  int *data_d, *output_d;
  int work_per_thrd;

  if (argc != 4) {
    printf("Correct way to execute this program is:\n");
    printf("./vectorOp data_size block_size work_per_thread\n");
    printf("For example:\n./vectorOp 10000 512 4\n");
    return 1;
  }

  data_size = atoi(argv[1]);
  block_size = atoi(argv[2]);
  work_per_thrd = atoi(argv[3]);

  // Initialize data on Host
  initialize_data_random(&data_h, data_size);
  initialize_data_zero(&output_h, data_size);
  initialize_data_zero(&device_output_h, data_size);

  // Initialize data on Device
  CUDA_CHECK_RETURN(hipMalloc((void**)&data_d, sizeof(int) * data_size));
  CUDA_CHECK_RETURN(hipMalloc((void**)&output_d, sizeof(int) * data_size));

  // Sequential vector operation
  set_clock();
  {
    sequential_vector_operation(output_h, data_h, data_size);
  }
  elapsed_time_CPU = get_elapsed_time();
  printf("-> Naive vector operation time: %.4fms\n", elapsed_time_CPU / 1000);

  // CUDA Parallel vector operation
  grid_size = (data_size - 1) / block_size * work_per_thrd + 1; // FIXME
  dim3 grid_dime(grid_size, 1, 1);
  dim3 block_dime(block_size, 1, 1);

  set_clock();
  {
    CUDA_CHECK_RETURN(hipMemcpy(
      data_d, data_h, sizeof(int) * data_size, hipMemcpyHostToDevice));

    // vector_operation_kernel<<<grid_dime, block_dime>>>(
    // output_d, data_d, work_per_thrd, data_size);

    vector_operation_kernel<<<grid_dime, block_dime>>>(
      output_d, data_d, data_size);

    // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    // Copy back the result
    CUDA_CHECK_RETURN(hipMemcpy(device_output_h,
                                 output_d,
                                 sizeof(int) * data_size,
                                 hipMemcpyDeviceToHost));
  }
  elapsed_time_CUDA = get_elapsed_time();

  printf("-> CUDA vector operation time: %.4fms\n", elapsed_time_CPU / 1000);
  printf("Achived speedup: %.4f\n", elapsed_time_CPU / elapsed_time_CUDA);

#ifdef TEST
  validate(output_h, device_output_h, data_size);
#endif

  free(data_h);
  free(output_h);
  free(device_output_h);

  CUDA_CHECK_RETURN(hipFree(output_d));
  CUDA_CHECK_RETURN(hipFree(data_d));

  return 0;
}
